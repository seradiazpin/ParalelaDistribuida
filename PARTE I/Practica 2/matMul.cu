#include <stdio.h>
#include<time.h>

#include "hip/hip_runtime.h"


__global__ void matrixMultGPU(int *a, int *b, int *c, int N, int particiones ) {
 int col = threadIdx.x + blockDim.x * blockIdx.x;

 int inicioParte = col*(particiones);
 int finParte = (col+1)*(particiones);

			for (int i = inicioParte; i < finParte; i++)
			{	
				for (int j = 0; j < N; j++)
				{
					int res = 0;
					for(int k=0;k<N;k++){
					    res =(res + a[i* N + k] * b[k * N + j]);
					    //printf("%i * %i +",A[i][k],B[k][j]);
					} 
					c[ i* N + j] = res;
				}
				
			}
  
 
}


int main(int argc, char* const argv[]) {

	if(argv[1]== NULL){
  		perror("Digite el tamaño de la matriz N");
  		exit(-1);
	}
	if(argv[2] == NULL){
		perror("Digite el numero de hilos ");
  		exit(-1);
	}
	if(argv[3] == NULL){
		perror("Digite el numero de Bloques ");
  		exit(-1);
	}
	int N = atoi(argv[1]);
	int nHilos = atoi(argv[2]);
	int bloq = atoi(argv[3]);
	printf("N = %i, numero hilos= %i\n", N, nHilos);
	if (N != 2 && N != 4 && N != 8 && N != 16 && N != 32 && N != 64 && N != 128 && N != 256 && N != 512 && N!= 1024)
	{
		perror("El tamaño debe ser un numero de estos [2,4,8,16,32,64,128, 256, 512, 1024]");
  		exit(-1);
	}
	if(nHilos > N){
		perror("Los hilos deben ser menores o iguales a el tamaño");
  		exit(-1);
	}
	if(bloq != 1 && bloq !=2 && bloq != 4 && bloq != 8 && bloq*nHilos > N){
		perror("Los bloques deben ser 1 2 4 8");
  		exit(-1);
	}
    
 int block = bloq ;
 int thread = nHilos;
 
    
 int *a = (int *)malloc(N*N*sizeof(int));

 int *b= (int *)malloc(N*N*sizeof(int));
	    
 int *c= (int *)malloc(N*N*sizeof(int));
    
 int *dev_a, *dev_b, *dev_c;
 int i,j;

  /* inicializando variables con datos*/
    
	for (i = 0; i<N; i++){
		for (j = 0; j<N; j++){
			a[(N*i)+j] = rand() & 0xf;
          
			b[(N*i)+j] = rand() & 0Xf;
		}
	}
    
  
  int size = N * N * sizeof(int);

 hipMalloc((void **) &dev_a, size);
 hipMalloc((void **) &dev_b, size);
 hipMalloc((void **) &dev_c, size);

 hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
 hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

 dim3 dimGrid(block);
 dim3 dimBlock(thread);

 int particiones = N/thread;
 clock_t start = clock();
 matrixMultGPU<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c,N, particiones);
    

 hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

 clock_t end = clock();
 float ans = (float)(end - start) / CLOCKS_PER_SEC;
 printf("Duracion hilos CUDA Tamaño %d: %.20f \n", N , ans);

 
 hipFree(dev_a);
 hipFree(dev_b);
 hipFree(dev_c);
 

  return 0;

}



// Using CUDA device to calculate pi
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define NUM_BLOCK  100  // Number of thread blocks
#define NUM_THREAD 1000  // Number of threads per block
#define nHilos 2
#define N 2


void imprimirMatriz(int row, int col, int **matrix){
	printf("--------------------------\n");
	printf("matrix %i X %i:\n",row,col);
	for(int i=0;i<row;i++){
        for(int j=0;j<col;j++){
            printf("%i ", matrix[i][j]);
        }
        printf("\n");
     }
}

void initMatriz(int row, int col, int **matrix){ 
     
     printf("matrix %i X %i:\n",row,col);

     for(int i=0;i<row;i++){
        for(int j=0;j<col;j++){
	        matrix[i][j]= rand() % 256;
	        //printf("%i ", matrix[i][j]);
        }
        //printf("\n");
     }
     printf("MATOK\n");
}


// Kernel that executes on the CUDA device
__global__ void matMul(int **M,int **M1, int **M2, int particiones) {

	int idx = blockIdx.x*blockDim.x+threadIdx.x;  // Sequential thread index across the blocks
    
	//printf("hilo->%i\n",hiloInfo->hil);
	int inicioParte = idx*(particiones);
	int finParte = (idx+1)*(particiones);
	//printf("inicioParte->%i, finParte->%i\n", inicioParte,finParte);
			for (int i = inicioParte; i < finParte; i++)
			{
				for (int j = 0; j < N; j++)
				{
					int res = 0;
					for(int k=0;k<N;k++){
					    res =(res +(M[i][k] * M1[k][j]));
					    //printf("%i * %i +",A[i][k],B[k][j]);
					} 
					M2[i][j] = res;
				}
			}
}

// Main routine that executes on the host
int main(void) {
	dim3 dimGrid(NUM_BLOCK,1,1);  // Grid dimensions
	dim3 dimBlock(NUM_THREAD,1,1);  // Block dimensions
	size_t size = N*N*sizeof(int);  //Tamaño del arreglo

    printf("HOLA\n");
	int **h_MATRIZA;
	int **h_MATRIZB;
	int **h_MATRIZC;


	h_MATRIZA = (int **)malloc(size);  //  Reservar memoria en device
	h_MATRIZB = (int **)malloc(size);
	h_MATRIZC = (int **)malloc(size);


	initMatriz(N,N,h_MATRIZA);
    imprimirMatriz(N,N,h_MATRIZA);
	initMatriz(N,N,h_MATRIZB);
    imprimirMatriz(N,N,h_MATRIZB);

	int **d_MATRIZA;
	int **d_MATRIZB;
	int **d_MATRIZC;

	
	d_MATRIZA = (int **)malloc(size);  //  Reservar memoria en device
	d_MATRIZB = (int **)malloc(size);
	d_MATRIZC = (int **)malloc(size);
	hipMalloc((void **) &d_MATRIZA, size);  //Reservar memoria en device
	hipMalloc((void **) &d_MATRIZB, size);
	hipMalloc((void **) &d_MATRIZC, size);
	// Inicializar en device
	hipMemset(d_MATRIZA, 0, size);
	hipMemset(d_MATRIZB, 0, size);
	hipMemset(d_MATRIZC, 0, size);

	hipMemcpy(d_MATRIZA, h_MATRIZA, size, hipMemcpyHostToDevice);
	hipMemcpy(d_MATRIZA, h_MATRIZA, size, hipMemcpyHostToDevice);
	int particiones = N/nHilos;
	// Calculo pi
	matMul <<<dimGrid, dimBlock>>> (d_MATRIZA, d_MATRIZB, d_MATRIZC,particiones); 
	// Sumar los elementos del arreglo
	hipMemcpy(h_MATRIZC, d_MATRIZC, size, hipMemcpyDeviceToHost);


	free(h_MATRIZA);
	free(h_MATRIZB); 
	free(h_MATRIZC); 
	hipFree(d_MATRIZA);
	hipFree(d_MATRIZB);
	hipFree(d_MATRIZC);

	return 0;
}


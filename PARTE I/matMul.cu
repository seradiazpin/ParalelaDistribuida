#include "hip/hip_runtime.h"
#include <unistd.h>
#include <stdio.h>
#include <errno.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define NUM_BLOCK  1  // Number of thread blocks
#define NUM_THREAD 1024 // Number of threads per block




void initMatriz(int row, int col, int **matrix){ 
     
     printf("matrix %i X %i:\n",row,col);

     for(int i=0;i<row;i++){
        for(int j=0;j<col;j++){
	        matrix[i][j]= rand() % 256;
	        //printf("%i ", matrix[i][j]);
        }
        //printf("\n");
     }
     printf("MATOK\n");
}



void imprimirMatriz(int row, int col, int **matrix){
	printf("--------------------------\n");
	printf("matrix %i X %i:\n",row,col);
	for(int i=0;i<row;i++){
        for(int j=0;j<col;j++){
            printf("%i ", matrix[i][j]);
        }
        printf("\n");
     }
}


__global__ void matrixMul(int N,int **M,int **M1,int **M2){
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    int particiones = N/NUM_THREAD;
	int inicioParte = idx*(particiones);
	int finParte = (idx+1)*(particiones);
    for (int i = inicioParte; i < finParte; i++)
			{
				for (int j = 0; j < N; j++)
				{
					int res = 0;
                    for(int k=0;k<N;k++){
                        res =(res +(M[fil][k]* M1[k][col]));
                        //printf("%i * %i +",A[i][k],B[k][j]);
                    } 
                    M2[fil][col] = res;

                    //printf("res -> %i\n",res );
					k++;
				}
			}
	

}





int main(int argc, char* const argv[]){

	if(argv[1]== NULL){
  		perror("Digite el tamaño de la matriz N");
  		exit(-1);
	}
	if(argv[2] == NULL){
		perror("Digite el numero de hilos ");
  		exit(-1);
	}
	N = atoi(argv[1]);
	nHilos = atoi(argv[2]);
	printf("N = %i, numero hilos= %i\n", N, nHilos);
	if (N != 2 && N != 4 && N != 8 && N != 16 && N != 32 && N != 64 && N != 128 && N != 256 && N != 512 && N!= 1024)
	{
		perror("El tamaño debe ser un numero de estos [2,4,8,16,32,64,128, 256, 512, 1024]");
  		exit(-1);
	}
	if(nHilos > N){
		perror("Los hilos deven ser menores o iguales a el tamaño");
  		exit(-1);
	}

	srand(time(NULL));

	int hilosErr;
	struct hilosInfo hilosArray[nHilos];
	pthread_t hilos [nHilos];


	
	int *MATRIZA[N];
	    for (int i=0; i<N; i++)
	         MATRIZA[i] = (int *)malloc(N * sizeof(int));

	initMatriz(N,N,MATRIZA);
	//imprimirMatriz(N,N,MATRIZA);

	int *MATRIZB[N];
	    for (int i=0; i<N; i++)
	         MATRIZB[i] = (int *)malloc(N * sizeof(int));
	    
	initMatriz(N,N,MATRIZB);
	//imprimirMatriz(N,N,MATRIZB);

	int *MATRIZC[N];
	    for (int i=0; i<N; i++)
	         MATRIZC[i] = (int *)malloc(N * sizeof(int));
	    
	printf("nhilos %i\n",nHilos );
	clock_t begin, end;
	double time_spent;
	begin = clock();

	for (int i = 0; i < nHilos; i++)
	{
		hilosArray[i].fila = N;
		hilosArray[i].columna = N;
		hilosArray[i].A = MATRIZA;
		hilosArray[i].B = MATRIZB;
		hilosArray[i].C = MATRIZC;
		hilosArray[i].hil = i;

		hilosErr= pthread_create(&hilos[i],NULL,test,&hilosArray[i]);
		if(hilosErr!=0){
			perror("ERROR CREAR HILO");
		}
	}
	for (int i = 0; i < nHilos; ++i)
	{
		hilosErr = pthread_join(hilos[i],NULL);
		if(hilosErr!=0){
			perror("ERROR CERRAR HILO");
		}
	}
	end = clock();
	time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

	printf("---------------\n");
	printf("Elapsed time: %.2lf seconds.\n", time_spent);
	//imprimirMatriz(N,N,MATRIZC);
}

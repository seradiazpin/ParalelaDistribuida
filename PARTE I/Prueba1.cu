#include <stdio.h>
#include<time.h>

#include "hip/hip_runtime.h"


__global__ void matrixMultGPU(int *a, int *b, int *c, int N, int particiones ) {
 int col = threadIdx.x + blockDim.x * blockIdx.x;

 int inicioParte = col*(particiones);
 int finParte = (col+1)*(particiones);
	//printf("inicioParte->%i, finParte->%i\n", inicioParte,finParte);
			for (int i = inicioParte; i < finParte; i++)
			{	
				for (int j = 0; j < N; j++)
				{
					int res = 0;
					for(int k=0;k<N;k++){
					    res =(res + a[i* N + k] * b[k * N + j]);
					    //printf("%i * %i +",A[i][k],B[k][j]);
					} 
					c[ i* N + j] = res;
				}
				
			}
  
 
}


int main(int argc, char* const argv[]) {

	if(argv[1]== NULL){
  		perror("Digite el tamaño de la matriz N");
  		exit(-1);
	}
	if(argv[2] == NULL){
		perror("Digite el numero de hilos ");
  		exit(-1);
	}
	if(argv[3] == NULL){
		perror("Digite el numero de Bloques ");
  		exit(-1);
	}
	int N = atoi(argv[1]);
	int nHilos = atoi(argv[2]);
	int bloq = atoi(argv[3]);
	printf("N = %i, numero hilos= %i\n", N, nHilos);
	if (N != 2 && N != 4 && N != 8 && N != 16 && N != 32 && N != 64 && N != 128 && N != 256 && N != 512 && N!= 1024)
	{
		perror("El tamaño debe ser un numero de estos [2,4,8,16,32,64,128, 256, 512, 1024]");
  		exit(-1);
	}
	if(nHilos > N){
		perror("Los hilos deben ser menores o iguales a el tamaño");
  		exit(-1);
	}
	if(bloq != 1 && bloq !=2 && bloq != 4 && bloq != 8 && bloq*nHilos > N){
		perror("Los bloques deben ser 1 2 4 8");
  		exit(-1);
	}
    
 int block = bloq ;
 int thread = nHilos;
 //srand(time(NULL));
    
 int *a = (int *)malloc(N*N*sizeof(int));

 int *b= (int *)malloc(N*N*sizeof(int));
	    
 int *c= (int *)malloc(N*N*sizeof(int));
    
 int *dev_a, *dev_b, *dev_c;
 int i,j;

  /* inicializando variables con datos*/
    printf("LOL\n");
	for (i = 0; i<N; i++){
		for (j = 0; j<N; j++){
			a[(N*i)+j] = rand() & 0xf;
          
			b[(N*i)+j] = rand() & 0Xf;
		}
	}
    /*
    for(int i = 0;i< N;i++){
        for(int j = 0;j< N;j++){
            printf("%i ",a[(N*i)+j]);
        }
            printf("\n");
    }
	*/
  
  int size = N * N * sizeof(int);

 hipMalloc((void **) &dev_a, size);
 hipMalloc((void **) &dev_b, size);
 hipMalloc((void **) &dev_c, size);

 hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
 hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

 dim3 dimGrid(block);
 dim3 dimBlock(thread);

 int particiones = N/thread;
 clock_t start = clock();
 matrixMultGPU<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c,N, particiones);
    

 hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

 clock_t end = clock();
 float ans = (float)(end - start) / CLOCKS_PER_SEC;
 printf("Duracion hilos CUDA Tamaño %d: %.20f \n", N , ans);

 /*
    printf("\nA \n");
    for(int i = 0;i< N;i++){
        for(int j = 0;j< N;j++){
            printf("%i ",a[(N*i)+j]);
        }
            printf("\n");
    }
	printf("\nB \n");
    for(int i = 0;i< N;i++){
        for(int j = 0;j< N;j++){
            printf("%i ",b[(N*i)+j]);
        }
            printf("\n");
    }
	printf("\nSolucion \n");
    for(int i = 0;i< N;i++){
        for(int j = 0;j< N;j++){
            printf("%i ",c[(N*i)+j]);
        }
            printf("\n");
    }
	*/
 hipFree(dev_a);
 hipFree(dev_b);
 hipFree(dev_c);
 

  return 0;

}

/*
9 14 1 6 2 9 1 9
3 11 7 14 4 13 6 7
3 13 3 11 15 10 9 5
13 11 6 5 3 9 12 14
0 0 0 6 2 2 14 9
13 4 4 6 11 0 2 1
6 1 1 15 0 9 9 13
12 14 9 9 2 9 3 7

3 4 12 14 0 1 11 11
6 12 12 9 14 12 7 14
2 8 11 6 3 13 8 15
4 12 5 11 13 10 11 2
10 14 8 13 9 6 13 12
15 2 0 0 4 7 11 2
2 13 11 10 10 2 2 9
0 5 5 0 10 12 15 6
*/

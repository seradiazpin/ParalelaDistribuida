#include <stdio.h>
#include<time.h>

#include "hip/hip_runtime.h"


__global__ void matrixMultGPU(int *a, int *b, int *c, int N) {
 int k, sum = 0;
 int col = threadIdx.x + blockDim.x * blockIdx.x;
 int fil = threadIdx.y + blockDim.y * blockIdx.y;

  if (col < N && fil < N) {
  for (k = 0; k < N; k++) {
   sum += a[fil * N + k] * b[k * N + col];
  }
  c[fil * N + col] = sum;
 }
}


int main() {
    
 const int N = 2;
 int block = 1;
 int thread = 2;
    
 int *a = (int *)malloc(N*N*sizeof(int));

 int *b= (int *)malloc(N*N*sizeof(int));
	    
 int *c= (int *)malloc(N*N*sizeof(int));
    
 int *dev_a, *dev_b, *dev_c;
 int i,j;

  /* inicializando variables con datos*/
    printf("LOL\n");
	for (i = 0; i<N; i++){
		for (j = 0; j<N; j++){
			a[(N*i)+j] = rand() & 0xf;
          
			b[(N*i)+j] = rand() & 0Xf;
		}
	}
    
    for(int i = 0;i< N;i++){
        for(int j = 0;j< N;j++){
            printf("%i ",a[(N*i)+j]);
        }
            printf("\n");
    }

  clock_t start = clock();
  int size = N * N * sizeof(int);

 hipMalloc((void **) &dev_a, size);
 hipMalloc((void **) &dev_b, size);
 hipMalloc((void **) &dev_c, size);

 hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
 hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

 dim3 dimGrid(block);
 dim3 dimBlock(thread);


 matrixMultGPU<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c,N);
    
for (i = 0; i<N; i++){
		for (j = 0; j<N; j++){
			a[(N*i)+j] = 0;
          
		}
	}
 hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

 clock_t end = clock();
 float ans = (float)(end - start) / CLOCKS_PER_SEC;
 printf("Duracion hilos CUDA Tamaño %d: %f \n", N , ans);

    
    for(int i = 0;i< N;i++){
        for(int j = 0;j< N;j++){
            printf("%i ",a[(N*i)+j]);
        }
            printf("\n");
    }
    for(int i = 0;i< N;i++){
        for(int j = 0;j< N;j++){
            printf("%i ",b[(N*i)+j]);
        }
            printf("\n");
    }
    for(int i = 0;i< N;i++){
        for(int j = 0;j< N;j++){
            printf("%i ",c[(N*i)+j]);
        }
            printf("\n");
    }
 hipFree(dev_a);
 hipFree(dev_b);
 hipFree(dev_c);


  return 0;

}

